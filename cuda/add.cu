
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int n, const float *a, const float *b, float *c)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

int main(int argc, char *argv[])
{
    float a[] = {1.0f, 2.0f, 3.0f, 4.0f, 5.0f};
    float b[] = {2.0f, 7.0f, -4.0f, 7.2f, 9.5f};
    const size_t n = 5;

    hipError_t err = hipSuccess;

    float *device_a = NULL;
    err = hipMalloc((void **)&device_a, sizeof(a));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (%s)\n", hipGetErrorString(err));
        return 1;
    }

    float *device_b = NULL;
    err = hipMalloc((void **)&device_b, sizeof(b));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (%s)\n", hipGetErrorString(err));
        return 1;
    }

    float *device_c = NULL;
    err = hipMalloc((void **)&device_c, n * sizeof(float));
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (%s)\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpy(device_a, a, sizeof(a), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from host to device (%s)\n", hipGetErrorString(err));
        return 1;
    }

    err = hipMemcpy(device_b, b, sizeof(b), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from host to device (%s)\n", hipGetErrorString(err));
        return 1;
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    add<<<blocksPerGrid, threadsPerBlock>>>(n, device_a, device_b, device_c);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch add kernel (%s)\n", hipGetErrorString(err));
        return 1;
    }

    float c[5];
    err = hipMemcpy(c, device_c, sizeof(c), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from device to host (%s)\n", hipGetErrorString(err));
        return 1;
    }

    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_c);

    for (size_t i = 0; i < n; ++i) {
        printf("c[%zu] = %g\n", i, c[i]);
    }
}
